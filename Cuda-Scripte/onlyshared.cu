#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cmath>
#include <time.h>
#include <iostream>

#define DTYPE float
__global__ void kernel(float *a, float *x, float* buff,int Xblocks,int size,bool comp,int toreduce)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
   	int j=threadIdx.y+blockIdx.y*blockDim.y;
   	__shared__ float xSM[1024];
   	if(comp==true){
		xSM[threadIdx.x+threadIdx.y*blockDim.x]=a[i+j*size]*x[i];
		__syncthreads();
	}
	else{
		xSM[threadIdx.x+threadIdx.y*blockDim.x]=buff[i+j*Xblocks];
	}   	
   	if(i<toreduce){	
        	for(int k=blockDim.x/2;k>0;k/=2){
            		if(threadIdx.x<k&&i+k<toreduce){
                		xSM[threadIdx.x+threadIdx.y*blockDim.x]+=xSM[threadIdx.x+k+threadIdx.y*blockDim.x];
                		__syncthreads();
            		}
        	}
        }
       if(threadIdx.x==0){
		}
       	buff[blockIdx.x+j*Xblocks]=xSM[threadIdx.y*blockDim.x];
}

void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++){
	x[i]=1;
//      x[i]= (DTYPE)(i+1);
	}
}
int main(int argc, char**argv)
{   
   int sx=32;
   int sy=32;
   int i=1;
   bool standard=true;
   if (argc>1)   
   {
	standard=false;
	sx=atoi(argv[1]);
      	if (argc>2){ 
      		sy=atoi(argv[2]);
      		if(argc>3){ 
      			i=atoi(argv[3]);      					
      		}
      	}	
   }
   if(standard){
   	std::cout<<"Do experiment with standard settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\nSy="<<sy<<"\nSize=1024*"<<i<<std::endl;
   }else 
   {
   	std::cout<<"Do experiment with individual settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\nSy="<<sy<<"\nSize=1024*"<<i<<std::endl;
       }
   /*if(sx*sy!=t){
   	std::cout<<"Sx*Sy has to be equal to threads per block"<<std::endl;
   	return -1;
   }*/
   int size=1024*i;
   int xblocks=size/sx;
   //Datenfelder anlegen für Host
   DTYPE *a_host, *buff_host,*x_host;
   //und Device
   DTYPE *a_dev, *buff_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   //htd: Host->Device Memcpy von A und x
   //dth: Device->Host Memcpy von y
   //kernelA, kernelAT
   float kernelA_time=0.0;

   //TODO: Host Speicher anlegen und A und x füllen
   a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
   x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
   buff_host=(DTYPE*)malloc(xblocks*size*sizeof(DTYPE));

   fillA(a_host,size);
   fillX(x_host,size);
   //TODO: CUDA Events erstellen

   //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
   hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
   hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
   hipMalloc((void**)&buff_dev,xblocks*size*sizeof(DTYPE));

	hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
	hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
   
   dim3 block(sx,sy);
   dim3 grid(size/block.x,size/block.y);
   
   int toreduce=size;//Anzahl der noch zu reduzierenden Wert, im ersten Kernel=size, da noch nichts reduziert wurde und man somit im ersten Berechnungsschritt size viel Skalare erhällt
   bool doComputation=true;

	//cache Konfiguration
	if(argc>4){
		if(atoi(argv[4])==1){//L1 Prefered
			std::cout<<"16 kB shared, 48kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
		} else if(atoi(argv[4])==2){
			std::cout<<"48kB shared, 16kb L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
		}else{
			std::cout<<"32kB shared, 32kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferNone);
		}
	}
   //Start Zeitmessung, da dies nur der erste Aufruf ist und danach der Kernel nochmal aufgerufen wird, mit einem anderen Argument als du Computation, endet die Zeitmessung erst nach der while Schleife, inder der Kernel noch einmal aufgerufen wird.
   hipEventCreate(&start);
   hipEventCreate(&end);
   hipEventRecord(start,0);
   kernel<<<grid,block>>>(a_dev,x_dev,buff_dev,xblocks,size,doComputation,toreduce);//,y_dev,size);

	//Überprüfung des Zwischenergebnisses
	/*cudaMemcpy(buff_host,buff_dev,xblocks*size*sizeof(DTYPE),cudaMemcpyDeviceToHost);
   for(int lj=0;lj<10;lj++){
   	for(int li=0;li<10;li++)
   		std::cout<<buff_host[li+xblocks*lj]<<" ; ";
   	std::cout<<std::endl;
   }*/
   //Vorbereitung für weitere Durchführung
   doComputation=false; //Matrix Vektor Multiplikation musst nur einmal durchgeführt werden
   if(toreduce%sx==0)
   	toreduce/=sx;
   else
   	toreduce=toreduce/sx+1;
   while(toreduce>1){//toreduce=1 bedeut, dass auf einen Wert reduziert wurde und somit keine weitere Reduzierung notwendig ist
   	int xgrid;
   	int ygrid=size/sy;
   	if(toreduce%sx==0)
   		xgrid=toreduce/sx;
   	else
	   	xgrid=toreduce/sx+1;
   	dim3 itgrid(xgrid,ygrid);
   	kernel<<<itgrid,block>>>(a_dev,x_dev,buff_dev,xblocks,size,doComputation,toreduce);//,y_dev,size);
   	if(toreduce%sx==0)
   	toreduce/=sx;
	else
   	toreduce=toreduce/sx+1;	
   }
   hipEventRecord(end,0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&kernelA_time,start,end);
   if(argc>4){
   	hipMemcpy(buff_host,buff_dev,xblocks*size*sizeof(DTYPE),hipMemcpyDeviceToHost);
   	std::cout<<"Das Endergebnis ist"<<std::endl;
   	for(int lj=0;lj<10;lj++){
   		//for(int li=0;li<10;li++)
   		//	std::cout<<buff_host[li+xblocks*lj]<<" ; ";
   //   		std::cout<<std::endl;
   		std::cout<<buff_host[xblocks*lj]<<std::endl;
   	}
   }
   std::cout<<"Computation time: "<<kernelA_time<<std::endl;
   //Bestimmung der Performance
   //float time_s=kernelA_time/1000;
   //float gflops=pow(10,-9)*size*size*2/time_s;
   float gflops=pow(10,-6)*size*size*2/kernelA_time;
   std::cout<<"Computation Performance in GFLOPs: "<<gflops<<std::endl;
   return 0;
}

