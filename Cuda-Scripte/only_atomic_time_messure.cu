#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>

#define DTYPE float
__global__ void kernel(float *a, float *x, float* buff,int Xblocks,int size)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
   	int j=threadIdx.y+blockIdx.y*blockDim.y;
   	if (i<size && j<size)
   {
	float addsc=a[i+j*size]*x[i];
      atomicAdd(&buff[j],addsc);
   }
}

void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++){
	x[i]=1;
//      x[i]= (DTYPE)(i+1);
	}
}
int main(int argc, char**argv)
{      
   int sx=32;
   int sy=32;
   int i=1;
   bool standard=true;
   if (argc>1)   
   {
	standard=false;
	sx=atoi(argv[1]);
      	if (argc>2){ 
      		sy=atoi(argv[2]);
      		if(argc>3){ 
      			i=atoi(argv[3]);      					
      		}
      	}	
   }
   if(standard){
   	std::cout<<"Do experiment with standard settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\nSy="<<sy<<"\nSize=1024*"<<i<<std::endl;
   }else 
   {
   	std::cout<<"Do experiment with individual settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\nSy="<<sy<<"\nSize=1024*"<<i<<std::endl;
       }
   /*if(sx*sy!=t){
   	std::cout<<"Sx*Sy has to be equal to threads per block"<<std::endl;
   	return -1;
   }*/
   int size=1024*i;
   int xblocks=size/sx;
   //Datenfelder anlegen für Host
   DTYPE *a_host, *buff_host, *x_host;
   //und Device
   DTYPE *a_dev, *buff_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   float kernelA_time=0.0;

   //TODO: Host Speicher anlegen und A und x füllen
   a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
   x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
   buff_host=(DTYPE*)malloc(size*sizeof(DTYPE));

   fillA(a_host,size);
   fillX(x_host,size);
   //TODO: CUDA Events erstellen

   //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
   hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
   hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
   hipMalloc((void**)&buff_dev,size*sizeof(DTYPE));

   //TODO: Host->Device Memcpy von A und x + Zeitmessung
   //cudaMemcpy(a_dev,a_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);
   //cudaMemcpy(x_dev,x_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);

	hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
	hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
	//cudaMemcpy(buff_dev,buff_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);
   //Konfiguration der CUDA Kernels
   
   dim3 block(sx,sy);
   dim3 grid(size/block.x,size/block.y);
   
	//cache Konfiguration
	if(argc>4){
		if(atoi(argv[4])==1){//L1 Prefered
			std::cout<<"16 kB shared, 48kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
		} else if(atoi(argv[4])==2){
			std::cout<<"48kB shared, 16kb L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
		}else{
			std::cout<<"32kB shared, 32kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferNone);
		}
	}
	
   //TODO: kernelAx ausführen und Zeit messen
   hipEventCreate(&start);
   hipEventCreate(&end);
   hipEventRecord(start,0);
   kernel<<<grid,block>>>(a_dev,x_dev,buff_dev,xblocks,size);//,y_dev,size);
   hipEventRecord(end,0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&kernelA_time,start,end);
   
   if(argc>5){
    hipMemcpy(buff_host,buff_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
    for(int lj=0;lj<10;lj++){
   		std::cout<<buff_host[lj]<<std::endl;
    }
   }
   std::cout<<"Computation time: "<<kernelA_time<<std::endl;
	float gflops=pow(10,-6)*size*size*2/kernelA_time;
   std::cout<<"Computation Performance in GFLOPs: "<<gflops<<std::endl;
   return 0;
}
