#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>

#define DTYPE float

__global__ void kernelmull(float *a, float *x, float* buff,int size)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
   	int j=threadIdx.y+blockIdx.y*blockDim.y;
	buff[i+j*size]=a[i+j*size]*x[i];
	__syncthreads();
}

#define WARP_SIZE 32
__global__ void kernelshuffle(float *a, float *x, float *b ,int size)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
   	int j=threadIdx.y+blockIdx.y*blockDim.y;
   	float res=0.0f;
    __shared__ float sm[32];
   if (threadIdx.x<WARP_SIZE)
      sm[threadIdx.x]=0;
   __syncthreads();
   if (j<size)
   {
      for (int k=i;k<size;k+=blockDim.x*gridDim.x)
      {
         res+=a[k+j*size]*x[k];
      }
      #pragma unroll
      for (int mask=WARP_SIZE/2;mask>0;mask>>=1)
      {
         res+=__shfl_down(res,mask);
      }
      if (threadIdx.x%WARP_SIZE==0)
      {
         sm[threadIdx.x/WARP_SIZE]=res;
      }
      __syncthreads();
      if (threadIdx.x<WARP_SIZE)
      {
         res=sm[threadIdx.x];
         #pragma unroll
         for (int mask=WARP_SIZE/2;mask>0;mask>>=1)
         {
            res+=__shfl_down(res,mask);
         }
      }
      if (threadIdx.x==0)
      {
         //atomicAdd(&b[j],res);
         b[j]=res;
      }
   }
}

void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++){
	x[i]=1;
//      x[i]= (DTYPE)(i+1);
	}
}
int main(int argc, char**argv)
{      
   int sx=32;
   int sy=32;
   int i=1;
   bool standard=true;
   if (argc>1)   
   {
	standard=false;
	sx=atoi(argv[1]);
      	if (argc>2){ 
      		sy=atoi(argv[2]);
      		if(argc>3){ 
      			i=atoi(argv[3]);      					
      		}
      	}	
   }
   if(standard){
   	std::cout<<"Do experiment with standard settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\nSy="<<sy<<"\nSize=1024*"<<i<<std::endl;
   }else 
   {
   	std::cout<<"Do experiment with individual settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\nSy="<<sy<<"\nSize=1024*"<<i<<std::endl;
       }
   /*if(sx*sy!=t){
   	std::cout<<"Sx*Sy has to be equal to threads per block"<<std::endl;
   	return -1;
   }*/
   int size=1024*i;
   int xblocks=size/sx;
   //Datenfelder anlegen für Host
   DTYPE *a_host, *buff_host, *x_host, *b_host;
   //und Device
   DTYPE *a_dev, *buff_dev,*x_dev,*b_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   float kernelA_time=0.0;

   //TODO: Host Speicher anlegen und A und x füllen
   a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
   x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
   buff_host=(DTYPE*)malloc(size*sizeof(DTYPE));
    b_host=(DTYPE*)malloc(size*sizeof(DTYPE));

   fillA(a_host,size);
   fillX(x_host,size);
   //TODO: CUDA Events erstellen

   //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
   hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
   hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
   hipMalloc((void**)&buff_dev,size*size*sizeof(DTYPE));
    hipMalloc((void**)&b_dev,size*sizeof(DTYPE));

   //TODO: Host->Device Memcpy von A und x + Zeitmessung
   //cudaMemcpy(a_dev,a_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);
   //cudaMemcpy(x_dev,x_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);

	hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
	hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
	//cudaMemcpy(buff_dev,buff_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);
   //Konfiguration der CUDA Kernels
   
   dim3 block;
   dim3 blockmull(sx,sy);
 
   dim3 grid;
    dim3 gridmull(size/sx,size/sy);
    block.x=128;
    block.y=1;
    grid.x=1;
    grid.y=size;
   
	//cache Konfiguration
	if(argc>4){
		if(atoi(argv[4])==1){//L1 Prefered
			std::cout<<"16 kB shared, 48kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelshuffle), hipFuncCachePreferL1);
		} else if(atoi(argv[4])==2){
			std::cout<<"48kB shared, 16kb L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelshuffle), hipFuncCachePreferShared);
		}else{
			std::cout<<"32kB shared, 32kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelshuffle), hipFuncCachePreferNone);
		}
	}
	
   //TODO: kernelAx ausführen und Zeit messen
   hipEventCreate(&start);
   hipEventCreate(&end);
   hipEventRecord(start,0);
    //kernelmull<<<gridmull,blockmull>>>(a_dev,x_dev,buff_dev,size);
    //cudaDeviceSynchronize();
   kernelshuffle<<<grid,block>>>(a_dev,x_dev,b_dev,size);//,y_dev,size);
    hipDeviceSynchronize();
   hipEventRecord(end,0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&kernelA_time,start,end);
   
   if(argc>5){
    hipMemcpy(b_host,b_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
    for(int lj=0;lj<10;lj++){
   		std::cout<<b_host[lj]<<std::endl;
    }
   }
   std::cout<<"Computation time: "<<kernelA_time<<std::endl;
	float gflops=pow(10,-6)*size*size*2/kernelA_time;
   std::cout<<"Computation Performance in GFLOPs: "<<gflops<<std::endl;
   return 0;
}
