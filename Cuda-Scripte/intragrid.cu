#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
namespace cg=cooperative_groups;

#define DTYPE float

__global__ void kernel(DTYPE *a, DTYPE *x, DTYPE* buff,int size, int numberBlocks){
    
    grid_group grid=this_grid();

    __shared__ DTYPE sm[1024];

    int lx=size/blockDim.x;
    if(size % blockDim.x != 0)
    	lx++;    
    int ly=size/(numberBlocks*blockDim.y);
    if(size % numberBlocks*blockDim.y != 0);
    	ly++;        
    for(int g=0;g<lx;g++){
        for(int h=0;h<ly;h++){
            int i=threadIdx.x+g*blockDim.x;
            int j=threadIdx.y+blockIdx.y*blockDim.y+h*blockDim.y*numberBlocks;
            if(i<size && j<size){
                sm[threadIdx.x+threadIdx.y*blockDim.x]=a[i+j*size]*x[i];
                __syncthreads();
                for (int k=blockDim.x/2;k>0;k/=2){
                    if (threadIdx.x<k)
                    sm[threadIdx.x+threadIdx.y*blockDim.x]+=sm[threadIdx.x+k+threadIdx.y*blockDim.x];
                    __syncthreads();
                }
                if (threadIdx.x==0){
                    buff[j]+=sm[threadIdx.y*blockDim.x];
                }
            }
        }
        grid.sync();                
     }
}


void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++){
	x[i]=1;
//      x[i]= (DTYPE)(i+1);
	}
}
int main(int argc, char**argv)
{   
   int sx=32;
   int sy=32;
   int i=1;
   bool standard=true;
   if (argc>1)   
   {
	standard=false;
	sx=atoi(argv[1]);
      	if (argc>2){ 
      		sy=atoi(argv[2]);
      		if(argc>3){ 
      			i=atoi(argv[3]);      					
      		}
      	}	
   }
   if(standard){
   	std::cout<<"Do experiment with standard settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\n Sy="<<sy<<"\n Size=1024*"<<i<<std::endl;
   }else 
   {
   	std::cout<<"Do experiment with individual settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\n Sy="<<sy<<"\n Size=1024*"<<i<<std::endl;
       }
   int size=1024*i;
   int xblocks=size/sx;
   //Datenfelder anlegen für Host
   DTYPE *a_host, *buff_host, *x_host;
   //und Device
   DTYPE *a_dev, *buff_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   float kernelA_time=0.0;

   //TODO: Host Speicher anlegen und A und x füllen
   a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
   x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
   buff_host=(DTYPE*)malloc(size*sizeof(DTYPE));

   fillA(a_host,size);
   fillX(x_host,size);
   //TODO: CUDA Events erstellen

   //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
   hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
   hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
   hipMalloc((void**)&buff_dev,size*sizeof(DTYPE));

   //TODO: Host->Device Memcpy von A und x + Zeitmessung
   //hipMemcpy(a_dev,a_host,1*sizeof(DTYPE),hipMemcpyHostToDevice);
   //hipMemcpy(x_dev,x_host,1*sizeof(DTYPE),hipMemcpyHostToDevice);

   hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
   hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
   
   dim3 block(sx,sy);
   dim3 grid(size/block.x,size/block.y);
   
	//cache Konfiguration
	if(argc>4){
		if(atoi(argv[4])==1){//L1 Prefered
			std::cout<<"16 kB shared, 48kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
		} else if(atoi(argv[4])==2){
			std::cout<<"48kB shared, 16kb L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
		}else{
			std::cout<<"32kB shared, 32kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferNone);
		}
	}
    grid.x=1;
    grid.y=size/block.y;
    if(size % block.y !=0)
    	grid.y++;
	
    int dev=0;
    hipGetDevice(&dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    int numBlocksPerSm;
    
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,kernel,sx,0);

    void *args[]={(void *)&a_dev,(void*)&x_dev,(void*)&buff_dev,(void*)&size,(void*)&numBlocksPerSm};
    grid.y=(int)ceilf(numBlocksPerSm);
	
   //TODO: kernelAx ausführen und Zeit messen
   hipEventCreate(&start);
   hipEventCreate(&end);
   hipEventRecord(start,0);
   hipLaunchCooperativeKernel((void*)kernel,grid,block,args,numBlocksPerSm); 
   hipDeviceSynchronize();
   hipEventRecord(end,0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&kernelA_time,start,end);

   if(argc>5){
    hipMemcpy(buff_host,buff_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
    for(int lj=0;lj<10;lj++){
   		std::cout<<buff_host[lj]<<std::endl;
    } 
   }
   std::cout<<"Computation time: "<<kernelA_time<<std::endl;  
	float gflops=pow(10,-6)*size*size*2/kernelA_time;
   std::cout<<"Computation Performance in GFLOPs: "<<gflops<<std::endl;
   return 0;
}
