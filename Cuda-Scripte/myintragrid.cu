#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
namespace cg=cooperative_groups;

#define DTYPE float

__global__ void kernel(DTYPE *a, DTYPE *x, DTYPE* buff,int size, int numberBlocks){
    
    grid_group grid=this_grid();

    __shared__ DTYPE sm[1024];

    int lx=size/(numberBlocks*blockDim.x);
    if(size % numberBlocks*blockDim.x != 0)
    	lx++;    
    int ly=size/(blockDim.y);
    if(size % blockDim.y != 0);
    	ly++;        
    
    for(int h=0;h<ly;h++){
    	for(int g=0;g<lx;g++){
            int i=threadIdx.x+blockIdx.x*blockDim.x+g*blockDim.x*numberBlocks;
            int j=threadIdx.y+h*blockDim.y;
            if(i<size && j<size){
                sm[threadIdx.x+threadIdx.y*blockDim.x]=a[i+j*size]*x[i];
                __syncthreads();
                for (int k=blockDim.x/2;k>0;k/=2){
                    if (threadIdx.x<k)
                    sm[threadIdx.x+threadIdx.y*blockDim.x]+=sm[threadIdx.x+k+threadIdx.y*blockDim.x];
                    __syncthreads();
                }
                if (threadIdx.x==0){
                    buff[j]+=sm[threadIdx.y*blockDim.x];
                }
            }
        }
        grid.sync();                
     }
}


void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++){
	x[i]=1;
//      x[i]= (DTYPE)(i+1);
	}
}
int main(int argc, char**argv)
{   
   int sx=32;
   int sy=32;
   int i=1;
   bool standard=true;
   if (argc>1)   
   {
	standard=false;
	sx=atoi(argv[1]);
      	if (argc>2){ 
      		sy=atoi(argv[2]);
      		if(argc>3){ 
      			i=atoi(argv[3]);      					
      		}
      	}	
   }
   if(standard){
   	std::cout<<"Do experiment with standard settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\n Sy="<<sy<<"\n Size=1024*"<<i<<std::endl;
   }else 
   {
   	std::cout<<"Do experiment with individual settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\n Sy="<<sy<<"\n Size=1024*"<<i<<std::endl;
       }
   int size=1024*i;
   int xblocks=size/sx;
   //Datenfelder anlegen für Host
   DTYPE *a_host, *buff_host, *x_host;
   //und Device
   DTYPE *a_dev, *buff_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   float kernelA_time=0.0;

   //TODO: Host Speicher anlegen und A und x füllen
   a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
   x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
   buff_host=(DTYPE*)malloc(size*sizeof(DTYPE));

   fillA(a_host,size);
   fillX(x_host,size);
   //TODO: CUDA Events erstellen

   //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
   hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
   hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
   hipMalloc((void**)&buff_dev,size*sizeof(DTYPE));

   //TODO: Host->Device Memcpy von A und x + Zeitmessung
   //hipMemcpy(a_dev,a_host,1*sizeof(DTYPE),hipMemcpyHostToDevice);
   //hipMemcpy(x_dev,x_host,1*sizeof(DTYPE),hipMemcpyHostToDevice);

   hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
   hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
   
   dim3 block(sx,sy);
   dim3 grid(size/block.x,size/block.y);
   
	//cache Konfiguration
	if(argc>4){
		if(atoi(argv[4])==1){//L1 Prefered
			std::cout<<"16 kB shared, 48kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
		} else if(atoi(argv[4])==2){
			std::cout<<"48kB shared, 16kb L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferShared);
		}else{
			std::cout<<"32kB shared, 32kB L1"<<std::endl;
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferNone);
		}
	}
    grid.y=1;
    grid.x=size/block.x;
    if(size % block.x !=0)
    	grid.x++;
	
    int dev=0;
    hipGetDevice(&dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    int numBlocksPerSm;
    
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm,kernel,sx,0);

    void *args[]={(void *)&a_dev,(void*)&x_dev,(void*)&buff_dev,(void*)&size,(void*)&numBlocksPerSm};
    grid.y=(int)ceilf(numBlocksPerSm);
	
   //TODO: kernelAx ausführen und Zeit messen
   hipEventCreate(&start);
   hipEventCreate(&end);
   hipEventRecord(start,0);
   hipLaunchCooperativeKernel((void*)kernel,grid,block,args,numBlocksPerSm); 
   hipDeviceSynchronize();
   hipEventRecord(end,0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&kernelA_time,start,end);

   if(argc>5){
    hipMemcpy(buff_host,buff_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
    for(int lj=size-10;lj<size;lj++){
   		std::cout<<buff_host[lj]<<std::endl;
    } 
   }
   std::cout<<"Computation time: "<<kernelA_time<<std::endl;  
	float gflops=pow(10,-6)*size*size*2/kernelA_time;
   std::cout<<"Computation Performance in GFLOPs: "<<gflops<<std::endl;
   return 0;
}
