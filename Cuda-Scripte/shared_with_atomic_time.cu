#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>

#define DTYPE float
__global__ void kernel(float *a, float *x, float* buff,int Xblocks,int size)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
   	int j=threadIdx.y+blockIdx.y*blockDim.y;
   	__shared__ float xSM[1024];
	xSM[threadIdx.x+threadIdx.y*blockDim.x]=a[i+j*size]*x[i];
	__syncthreads();   	
   	if(i<size){	
        	for(int k=blockDim.x/2;k>0;k/=2){
            		if(threadIdx.x<k&&i+k<size){
                		xSM[threadIdx.x+threadIdx.y*blockDim.x]+=xSM[threadIdx.x+k+threadIdx.y*blockDim.x];
                		__syncthreads();
            		}
        	}
        }
	if (threadIdx.x==0){
         atomicAdd(&buff[j],xSM[threadIdx.y*blockDim.x]);
   	}
}

void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++){
	x[i]=1;
//      x[i]= (DTYPE)(i+1);
	}
}
int main(int argc, char**argv)
{   
   int sx=32;
   int sy=32;
   int i=1;
   bool standard=true;
   if (argc>1)   
   {
	standard=false;
	sx=atoi(argv[1]);
      	if (argc>2){ 
      		sy=atoi(argv[2]);
      		if(argc>3){ 
      			i=atoi(argv[3]);      					
      		}
      	}	
   }
   if(standard){
   	std::cout<<"Do experiment with standard settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\n Sy="<<sy<<"\n Size=1024*"<<i<<std::endl;
   }else 
   {
   	std::cout<<"Do experiment with individual settings"<<std::endl;
   	std::cout<<"Sx="<<sx<<"\n Sy="<<sy<<"\n Size=1024*"<<i<<std::endl;
       }
   /*if(sx*sy!=t){
   	std::cout<<"Sx*Sy has to be equal to threads per block"<<std::endl;
   	return -1;
   }*/
   int size=1024*i;
   int xblocks=size/sx;
   //Datenfelder anlegen für Host
   DTYPE *a_host, *buff_host, *x_host;
   //und Device
   DTYPE *a_dev, *buff_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   float kernelA_time=0.0;

   //TODO: Host Speicher anlegen und A und x füllen
   a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
   x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
   buff_host=(DTYPE*)malloc(size*sizeof(DTYPE));

   fillA(a_host,size);
   fillX(x_host,size);
   //TODO: CUDA Events erstellen

   //TODO: CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
   hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
   hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
   hipMalloc((void**)&buff_dev,size*sizeof(DTYPE));

   //TODO: Host->Device Memcpy von A und x + Zeitmessung
   //cudaMemcpy(a_dev,a_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);
   //cudaMemcpy(x_dev,x_host,1*sizeof(DTYPE),cudaMemcpyHostToDevice);

   hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
   hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
   
   dim3 block(sx,sy);
   dim3 grid(size/block.x,size/block.y);
   
   //TODO: kernelAx ausführen und Zeit messen
   hipEventCreate(&start);
   hipEventCreate(&end);
   hipEventRecord(start,0);
   kernel<<<grid,block>>>(a_dev,x_dev,buff_dev,xblocks,size);
   hipEventRecord(end,0);
   hipEventSynchronize(end);
   hipEventElapsedTime(&kernelA_time,start,end);

   if(argc>4){
    hipMemcpy(buff_host,buff_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
    for(int lj=0;lj<10;lj++){
   		std::cout<<buff_host[lj]<<std::endl;
    } 
   }
   std::cout<<"Computation time: "<<kernelA_time<<std::endl;  
	float gflops=pow(10,-6)*size*size*2/kernelA_time;
   std::cout<<"Computation Performance in GFLOPs: "<<gflops<<std::endl;
   return 0;
}
